#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdbool.h>

#define INITIAL_LINE_CAPACITY 100
#define LINE_INCREMENT 100

bool decode(const char *image, const char *passphrase) {
    char command[200];
    sprintf(command, "steghide extract -sf %s -p %s", image, passphrase);

    if (system(command) == 0) {
        printf("Data extracted successfully!\n");
        printf("Password: %s\n", passphrase);
        return true;
    } else {
        printf("Failed: %s\n", passphrase);
        return false;
    }
}

char **getLines(const char *passwordList, int *lineCount) {
    FILE *file = fopen(passwordList, "r");
    if (file == NULL) {
        printf("File not found!\n");
        exit(1);
    }

    int capacity = INITIAL_LINE_CAPACITY;
    char **lines = (char **)malloc(capacity * sizeof(char *));
    if (lines == NULL) {
        printf("Memory allocation failed!\n");
        exit(1);
    }

    *lineCount = 0;
    char buffer[200];
    while (fgets(buffer, sizeof(buffer), file) != NULL) {
        // Remove newline character if present
        buffer[strcspn(buffer, "\n")] = '\0';

        // Allocate memory for the line and copy the string
        lines[*lineCount] = (char *)malloc(strlen(buffer) + 1);
        if (lines[*lineCount] == NULL) {
            printf("Memory allocation failed!\n");
            exit(1);
        }
        strcpy(lines[*lineCount], buffer);

        (*lineCount)++;

        // Check if we need to resize the lines array
        if (*lineCount >= capacity) {
            capacity += LINE_INCREMENT;
            char **temp = (char **)realloc(lines, capacity * sizeof(char *));
            if (temp == NULL) {
                printf("Memory allocation failed!\n");
                exit(1);
            }
            lines = temp;
        }
    }

    fclose(file);
    return lines;
}

void freeLines(char **lines, int lineCount) {
    for (int i = 0; i < lineCount; i++) {
        free(lines[i]);
    }
    free(lines);
}

void bruteForce(const char *image, const char *passwordList) {
    int lineCount;
    char **lines = getLines(passwordList, &lineCount);
    for (int i = 0; i < lineCount; i++) {
        if (decode(image, lines[i])) {
            break;
        }
        printf("%d / %d\n", i+1, lineCount);
    }

    freeLines(lines, lineCount);
}

int main(int argc, char *argv[]) {
    printf("stegbreaker\n");

    if (argc != 3) {
        printf("Usage: %s <steg.jpg> <wordlist>\n", argv[0]);
        return 1;
    }

    const char *image = argv[1];
    const char *passwordList = argv[2];
    bruteForce(image, passwordList);

    return 0;
}
